#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <mma.h>

using namespace nvcuda;

#define WARP_SIZE 32
#define DEVICE_INLINE __device__ inline
#define HOST_DEVICE_INLINE __device__ __host__ inline
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2*>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162*>(&(value))[0])
#define LDST32BITS(value) (reinterpret_cast<half2*>(&(value))[0])
#define LDST64BITS(value) (reinterpret_cast<float2*>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4*>(&(value))[0])
#define CP_ASYNC_COMMIT_GROUP() asm volatile("cp.async.commit_group;\n" ::)
#define CP_ASYNC_WAIT_ALL() asm volatile("cp.async.wait_all;\n" ::)
#define CP_ASYNC_WAIT_GROUP(n) asm volatile("cp.async.wait_group %0;\n" ::"n"(n))
// ca(cache all, L1 + L2): support 4, 8, 16 bytes, cg(cache global, L2): only support 16 bytes.
#define CP_ASYNC_CA(dst, src, bytes) asm volatile("cp.async.ca.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))
#define CP_ASYNC_CG(dst, src, bytes) asm volatile("cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))
#define LDMATRIX_X1(R, addr) asm volatile("ldmatrix.sync.aligned.x1.m8n8.shared.b16 {%0}, [%1];\n" : "=r"(R) : "r"(addr))
#define LDMATRIX_X2(R0, R1, addr) asm volatile("ldmatrix.sync.aligned.x2.m8n8.shared.b16 {%0, %1}, [%2];\n" : "=r"(R0), "=r"(R1) : "r"(addr))
#define LDMATRIX_X4(R0, R1, R2, R3, addr) asm volatile("ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n" : "=r"(R0), "=r"(R1), "=r"(R2), "=r"(R3) : "r"(addr))
#define LDMATRIX_X1_T(R, addr) asm volatile("ldmatrix.sync.aligned.x1.trans.m8n8.shared.b16 {%0}, [%1];\n" : "=r"(R) : "r"(addr))
#define LDMATRIX_X2_T(R0, R1, addr) asm volatile("ldmatrix.sync.aligned.x2.trans.m8n8.shared.b16 {%0, %1}, [%2];\n" : "=r"(R0), "=r"(R1) : "r"(addr))
#define LDMATRIX_X4_T(R0, R1, R2, R3, addr) asm volatile("ldmatrix.sync.aligned.x4.trans.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n" : "=r"(R0), "=r"(R1), "=r"(R2), "=r"(R3) : "r"(addr))
#define HMMA16816(RD0, RD1, RA0, RA1, RA2, RA3, RB0, RB1, RC0, RC1) asm volatile("mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16 {%0, %1}, {%2, %3, %4, %5}, {%6, %7}, {%8, %9};\n" : "=r"(RD0), "=r"(RD1) : "r"(RA0), "r"(RA1), "r"(RA2), "r"(RA3), "r"(RB0), "r"(RB1), "r"(RC0), "r"(RC1))

HOST_DEVICE_INLINE 
int div_ceil(int a, int b) {return (a + b - 1) / b;}

// i: row index; j: col index
__device__ __host__ __forceinline__ int swizzle_j(int i, int j) {
    // s_a [16][16]的 half数据, blocksize=32, 每个线程8个数据
    // 每行16*2个字节，即8个bank, 每四行会有一次bank冲突

    // >>> sw(0,0),sw(0,8),sw(1,0),sw(1,8),sw(2,0),sw(2,8),sw(3,0),sw(3,8)       
    // (0, 8, 0, 8, 0, 8, 0, 8)
    // >>> sw(4,0),sw(4,8),sw(5,0),sw(5,8),sw(6,0),sw(6,8),sw(7,0),sw(7,8)       
    // (8, 0, 8, 0, 8, 0, 8, 0)
    // >>> sw(8,0),sw(8,8),sw(9,0),sw(9,8),sw(10,0),sw(10,8),sw(11,0),sw(11,8)       
    // (0, 8, 0, 8, 0, 8, 0, 8)
    // >>> sw(12,0),sw(12,8),sw(13,0),sw(13,8),sw(14,0),sw(14,8),sw(15,0),sw(15,8)       
    // (8, 0, 8, 0, 8, 0, 8, 0)
    return ((int(j / 8) ^ int(i / 4)) % 2) * 8;
}


void cpu_gemm_reference(const half* A, const half* B, half* C, int M, int N, int K) {
    for (int m = 0; m < M; m++) {
        for (int n = 0; n < N; n++) {
            float acc = 0.0f;
            for (int k = 0; k < K; k++) {
                acc += __half2float(A[m * K + k]) * __half2float(B[k * N + n]);
            }
            C[m * N + n] = __float2half(acc);
        }
    }
}


int check_results(const half* result, const half* reference, int M, int N, float threshold = 0.01f) {
    int error_count = 0;
    int max_errors_to_print = 10; // 只打印前10个错误
    int printed_errors = 0;
    
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            int idx = i * N + j;
            float res = __half2float(result[idx]);
            float ref = __half2float(reference[idx]);
            float abs_error = fabs(res - ref);
            float rel_error = (ref != 0.0f) ? abs_error / fabs(ref) : abs_error;
            
            if (abs_error > threshold) {
                error_count++;
                if (printed_errors < max_errors_to_print) {
                    printf("错误[%d,%d]: GPU=%.6f, CPU=%.6f, 绝对误差=%.6f, 相对误差=%.6f\n", 
                           i, j, res, ref, abs_error, rel_error);
                    printed_errors++;
                }
            }
        }
    }
    
    if (error_count > 0) {
        if (error_count > max_errors_to_print) {
            printf("... 共有 %d 个元素误差超过阈值 (只显示前 %d 个)\n", 
                   error_count, max_errors_to_print);
        }
    } else {
        printf("所有元素误差均在阈值内，验证通过！\n");
    }
    
    return error_count;
}


template<const int MMA_M=16, const int MMA_N=8, const int MMA_K=16>
__global__ void mma_simple_kernel(
    half* A, half* B, half* C, int M, int N, int K) {
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int NUM_K_TILES = div_ceil(K, MMA_K);
    constexpr int BM = MMA_M; // 16
    constexpr int BN = MMA_N; // 8
    constexpr int BK = MMA_K; // 16

    __shared__ half s_a[MMA_M][MMA_K]; // 16x16
    __shared__ half s_b[MMA_K][MMA_N]; // 16x8

    const int tid = threadIdx.y * blockDim.x + threadIdx.x; // within block
    const int lane_id = tid % WARP_SIZE; // 0~31

    // 单线程单指令一次最多读取16 bytes(128 bits)的数据，我们使用float4来读取
    // 当同一个warp内的32个线程访问连续地址时，CUDA硬件会自动将这些请求合并为更少的内存事务
    // 理想情况下，合并后的访问可以达到cache line的大小 128 bytes
    // 如下所示，我们控制每个线程访问连续的8个fp16数据

    // s_a[16][16], 每行16，每线程load 8，需要2线程，共16行，需2x16=32线程
    const int load_smem_a_m = tid / 2; // row 0~15
    const int load_smem_a_k = (tid % 2) * 8; // col 0,8
    // s_b[16][8], 每行8，每线程load 8，需要1线程，共16行，需16线程，只需一半线程加载
    const int load_smem_b_k = tid; // row 0~31, but only use 0~15
    const int load_smem_b_n = 0; // col 0
    const int load_gmem_a_m = by * BM + load_smem_a_m; // global m
    const int load_gmem_b_n = bx * BN + load_smem_b_n; // global n
    if (load_gmem_a_m >= M && load_gmem_b_n >= N) return;

    uint32_t RC[2] = {0, 0};

    #pragma unroll
    for (int k = 0; k < NUM_K_TILES; ++k) {
        // gmem_a -> smem_a
        int load_gmem_a_k = k * BK + load_smem_a_k; // global col of a
        int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
        LDST128BITS(s_a[load_smem_a_m][load_smem_a_k]) = (
            LDST128BITS(A[load_gmem_a_addr]));
        // LDST128BITS(s_a[load_smem_a_m][swizzle_j(
        //   load_smem_a_m, load_smem_a_k)]) = (LDST128BITS(A[load_gmem_a_addr]));

        // gmem_b -> smem_b
        if (lane_id < MMA_K) {
            int load_gmem_b_k = k * MMA_K + load_smem_b_k; // global row of b
            int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n; 
            LDST128BITS(s_b[load_smem_b_k][load_smem_b_n]) = (
                LDST128BITS(B[load_gmem_b_addr]));
        }
        // 加载后的寄存器数据排布会经过隐式重组，
        // 以匹配 mma.sync.aligned.m16n8k16
        __syncthreads();

        uint32_t RA[4];
        uint32_t RB[2];
        
        // ldmatrix for s_a, ldmatrix.trans for s_b.
        // s_a: [(0~15),(0,8)]
        uint32_t load_smem_a_ptr = __cvta_generic_to_shared(
            &s_a[lane_id % 16][(lane_id / 16) * 8]); 
        // uint32_t load_smem_a_ptr = __cvta_generic_to_shared(
        //   &s_a[lane_id % 16][swizzle_j(lane_id % 16, (lane_id / 16) * 8)]); 
        LDMATRIX_X4(RA[0], RA[1], RA[2], RA[3], load_smem_a_ptr);
        uint32_t load_smem_b_ptr = __cvta_generic_to_shared(
            &s_b[lane_id % 16][0]);
        LDMATRIX_X2_T(RB[0], RB[1], load_smem_b_ptr);

        HMMA16816(RC[0], RC[1], RA[0], RA[1], RA[2], RA[3], RB[0], RB[1], RC[0], RC[1]);

        __syncthreads();
    }
    
    // s_c[16][8], https://docs.nvidia.com/cuda/parallel-thread-execution/index.html
    // #matrix-fragments-for-mma-m16n8k16-with-floating-point-type
    // [0~7][0~3 u32 -> 0~7 f16], [8~15][0~3 u32 -> 0~7 f16]
    int store_lane_gmem_c_m = by * BM + lane_id / 4;
    int store_lane_gmem_c_n = bx * BN + (lane_id % 4) * 2;
    int store_gmem_c_addr_0 = store_lane_gmem_c_m * N + store_lane_gmem_c_n;
    int store_gmem_c_addr_1 = (store_lane_gmem_c_m + 8) * N + store_lane_gmem_c_n;
    LDST32BITS(C[store_gmem_c_addr_0]) = LDST32BITS(RC[0]); 
    LDST32BITS(C[store_gmem_c_addr_1]) = LDST32BITS(RC[1]); 
}

int main(int argc, char *argv[]) {
    int M = 16;
    int N = 8;
    int K = 16;
    if (argc > 1) M = std::stoi(argv[1]);
    if (argc > 2) N = std::stoi(argv[2]);
    if (argc > 3) K = std::stoi(argv[3]);
  
    size_t size_a = M * K * sizeof(half);
    size_t size_b = K * N * sizeof(half);
    size_t size_c = M * N * sizeof(half);
  
    half *h_a, *h_b, *h_c, *h_ref;
    half *d_a, *d_b, *d_c;
    h_a = (half *)malloc(size_a);
    h_b = (half *)malloc(size_b);
    h_c = (half *)malloc(size_c);
    h_ref = (half *)malloc(size_c);
  
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);
  
    // 初始化输入数据
    for (int i = 0; i < M * K; i++)
        h_a[i] = __float2half((float)(i % 16) / 10.0f);
    for (int i = 0; i < K * N; i++)
        h_b[i] = __float2half((float)(i % 8) / 10.0f);
    
    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);
    hipMemset(d_c, 0, size_c);
  
    // 设置CUDA内核参数
    constexpr int MMA_M = 16;
    constexpr int MMA_N = 8;
    constexpr int MMA_K = 16;   
    dim3 block(WARP_SIZE);
    dim3 grid(div_ceil(N, MMA_N), div_ceil(M, MMA_M));
  
    mma_simple_kernel<MMA_M, MMA_N, MMA_K><<<grid, block>>>(
        d_a, d_b, d_c, M, N, K
    );
    
    hipMemcpy(h_c, d_c, size_c, hipMemcpyDeviceToHost);
    
    cpu_gemm_reference(h_a, h_b, h_ref, M, N, K);
    
    float threshold = 0.01f; // 误差阈值
    int error_count = check_results(h_c, h_ref, M, N, threshold);
  
    // 释放资源
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);
    free(h_ref);
  
    return 0;
}

// nvcc -o simple_mma_fp16 simple_mma_fp16.cu -arch=sm_90